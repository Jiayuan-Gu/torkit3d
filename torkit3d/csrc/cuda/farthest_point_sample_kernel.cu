#include "hip/hip_runtime.h"
// CUDA Implementation for farthest point sampling.

// AT_ASSERT has become AT_CHECK on master after 0.4.
// AT_CHECK has become TORCH_CHEC  K on master after 1.2.
// CHECK_EQ, CHECK_GT, etc. are marcos in Pytorch (include ATen.h).
// Tensor.type() is deprecated and instead use Tensor.options() after 1.5.
// Tensor.data() is deprecated and instead use Tensor.data_ptr() after 1.5.

#include <algorithm>

#include <ATen/ATen.h>
#include <THC/THC.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
  CHECK_CUDA(x);       \
  CHECK_CONTIGUOUS(x)
// #define CHECK_EQ(x, y) TORCH_CHECK(x == y, #x " does not equal to " #y)
// #define CHECK_GT(x, y) TORCH_CHECK(x > y, #x " is not greater than " #y)

#define MAX_THREADS 512
inline int opt_n_threads(int work_size)
{
  const int pow_2 = std::log(static_cast<double>(work_size)) / std::log(2.0);
  return std::max(std::min(1 << pow_2, MAX_THREADS), 1);
}

/**
 * FPS kernel
 * points: [B, N1, D]
 * temp: [B, N1]
 * index: [B, N2]
 **/
template <unsigned int BLOCK_SIZE, unsigned int DIM, typename scalar_t, typename index_t>
__global__ void farthest_point_sample_kernel(
    index_t *__restrict__ index,
    const scalar_t *__restrict__ points,
    scalar_t *__restrict__ temp,
    const int64_t num_points,
    const int64_t num_samples)
{
  // Allocate shared memory
  __shared__ scalar_t smem_dist[BLOCK_SIZE];
  // Use int to save memory
  __shared__ int smem_idx[BLOCK_SIZE];

  const int batch_idx = blockIdx.x;
  int cur_idx = 0;
  int points_offset = batch_idx * num_points * DIM;
  int temp_offset = batch_idx * num_points;
  int index_offset = batch_idx * num_samples;

  // Explicitly choose the first point as a centroid
  if (threadIdx.x == 0)
    index[index_offset] = cur_idx;

  for (int i = 1; i < num_samples; ++i)
  {
    scalar_t max_dist = 0.0;
    int max_idx = cur_idx;

    int offset1 = cur_idx * DIM;
    scalar_t coords1[DIM] = {0.0};
#pragma unroll
    for (int ii = 0; ii < DIM; ++ii)
    {
      coords1[ii] = points[points_offset + offset1 + ii];
    }

    for (int j = threadIdx.x; j < num_points; j += BLOCK_SIZE)
    {
      int offset2 = j * DIM;
      scalar_t dist = 0.0;
#pragma unroll
      for (int jj = 0; jj < DIM; ++jj)
      {
        scalar_t diff = points[points_offset + offset2 + jj] - coords1[jj];
        dist += diff * diff;
      }

      scalar_t last_dist = temp[temp_offset + j];
      if (last_dist > dist || last_dist < 0.0)
      {
        temp[temp_offset + j] = dist;
      }
      else
      {
        dist = last_dist;
      }
      if (dist > max_dist)
      {
        max_dist = dist;
        max_idx = j;
      }
    }

    smem_dist[threadIdx.x] = max_dist;
    smem_idx[threadIdx.x] = max_idx;

    // assert block_size == blockDim.x
    int offset = BLOCK_SIZE / 2;
    while (offset > 0)
    {
      __syncthreads();
      if (threadIdx.x < offset)
      {
        scalar_t dist1 = smem_dist[threadIdx.x];
        scalar_t dist2 = smem_dist[threadIdx.x + offset];
        if (dist1 < dist2)
        {
          smem_dist[threadIdx.x] = dist2;
          smem_idx[threadIdx.x] = smem_idx[threadIdx.x + offset];
        }
      }
      offset /= 2;
    }
    __syncthreads();

    cur_idx = smem_idx[0];
    if (threadIdx.x == 0)
      index[index_offset + i] = (index_t)cur_idx;
  }
}

/**
 * Forward
 * Input:
 *  points: [B, N1, D]
 * Output:
 *  index: [B, N2]
 **/
at::Tensor farthest_point_sample_cuda(
    const at::Tensor points,
    const int64_t num_samples)
{

  // Sanity check
  CHECK_INPUT(points);
  CHECK_EQ(points.dim(), 3);
  CHECK_EQ(points.size(2), 3);
  CHECK_GT(num_samples, 0);
  CHECK_GE(points.size(1), num_samples);

  const auto batch_size = points.size(0);
  const auto num_points = points.size(1);
  const auto dim = points.size(2);

  auto index = at::zeros({batch_size, num_samples}, points.options().dtype(at::kLong));
  // In original implementation, it only allocates memory with the size of grid instead of batch size.
  auto temp = at::neg(at::ones({batch_size, num_points}, points.options()));

  // In order to make full use of shared memory and threads,
  // it is recommended to set num_samples to be power of 2.
  const auto n_threads = opt_n_threads(num_points);

#define RUN(BLOCK_SIZE, DIM)                                                                    \
  AT_DISPATCH_FLOATING_TYPES(points.scalar_type(), "farthest_point_sample_cuda", ([&] {         \
                               farthest_point_sample_kernel<BLOCK_SIZE, DIM, scalar_t, int64_t> \
                                   <<<batch_size, BLOCK_SIZE>>>(                                \
                                       index.data_ptr<int64_t>(),                               \
                                       points.data_ptr<scalar_t>(),                             \
                                       temp.data_ptr<scalar_t>(),                               \
                                       num_points,                                              \
                                       num_samples);                                            \
                             }));

#define CASE(BLOCK_SIZE) \
  case BLOCK_SIZE:       \
    RUN(BLOCK_SIZE, 3)   \
    break;

  switch (n_threads)
  {
    CASE(512)
    CASE(256)
    CASE(128)
    CASE(64)
    CASE(32)
    CASE(16)
  default:
    RUN(16, 3)
  }

  THCudaCheck(hipGetLastError());

  return index;
}